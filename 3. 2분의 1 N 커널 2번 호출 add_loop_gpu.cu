#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N 10000

__global__ void add(int *a, int *b, int *c, int start) {
    int tid = blockIdx.x; 
    if (tid < N / 2) {
        c[start + tid] = a[start + tid] + b[start + tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    clock_t start = clock();
    
    // 첫 번째 커널 실행
    add<<<N / 2, 1>>>(dev_a, dev_b, dev_c, 0);
    hipDeviceSynchronize();  

    // 두 번째 커널 실행
    add<<<N / 2, 1>>>(dev_a, dev_b, dev_c, N / 2);
    hipDeviceSynchronize();  
    
    clock_t end = clock();
    
    printf("소요시간: %lf 초\n", (double)(end - start) / CLOCKS_PER_SEC);
    
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++) { 
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    return 0;
}
