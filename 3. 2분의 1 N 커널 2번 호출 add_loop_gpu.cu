#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N 10000  // N을 10000으로 설정

__global__ void add(int *a, int *b, int *c, int start) {
    int tid = blockIdx.x;  // 블록 인덱스를 사용
    if (tid < N / 2) {
        c[start + tid] = a[start + tid] + b[start + tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    clock_t start = clock();
    
    // 첫 번째 Kernel 실행 (N/2 개만 처리)
    add<<<N / 2, 1>>>(dev_a, dev_b, dev_c, 0);
    hipDeviceSynchronize();  // 동기화

    // 두 번째 Kernel 실행 (나머지 N/2 개 처리)
    add<<<N / 2, 1>>>(dev_a, dev_b, dev_c, N / 2);
    hipDeviceSynchronize();  
    
    clock_t end = clock();
    
    printf("소요시간: %lf 초\n", (double)(end - start) / CLOCKS_PER_SEC);
    
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++) { 
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    return 0;
}
