#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N 10
__global__ void add( int *a, int *b, int *c) {
	int tid=blockIdx.x;
	if(tid<N)
		c[tid]= a[tid] + b[tid];
}

int main(void){
	int a[N] , b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	HANDLE_ERROR( hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR( hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR( hipMalloc((void**)&dev_c, N*sizeof(int)));

	for(int i=0; i<N; i++){
		a[i]=-i;
		b[i]=i*i;
	}

	HANDLE_ERROR( hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));
	clock_t start = clock();
	add<<<N, 1>>>(dev_a, dev_b, dev_c);
	clock_t end = clock();

	printf("소요시간: %lf\n",(double)(end-start)/CLOCKS_PER_SEC);
	HANDLE_ERROR( hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));
	for (int i=0; i<N; i++){
		printf("%d +%d = %d\n", a[i], b[i], c[i]);
	}
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
	return 0;
}



